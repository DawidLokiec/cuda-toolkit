#include "memory.cuh"
#include "error.cuh"
#include <stdexcept>

namespace {
	__host__ void copyDataBetweenCpuAndGpu(
			void *destinationGpuMemory,
			const void *sourceCpuMemory,
			const size_t numBytesToCopy,
			const hipMemcpyKind copyDirection
	) {
		const hipError_t status = hipMemcpy(destinationGpuMemory, sourceCpuMemory, numBytesToCopy, copyDirection);
		if (status != hipSuccess) {
			if (copyDirection == hipMemcpyHostToDevice) {
				throw std::runtime_error(
						"copyDataFromCpuMemoryToGpuMemory call failed with the CUDA error code " +
						CudaUtils::toErrorDescription(status)
				);
			} else {
				throw std::runtime_error(
						"copyDataFromGpuMemoryToCpuMemory call failed with the CUDA error code " +
						CudaUtils::toErrorDescription(status)
				);
			}
		}
	}
}

[[maybe_unused]] __host__ void CudaUtils::allocateGpuMemory(
		void **pointerToAllocatedMemory,
		const size_t memorySizeInBytes
) {
	const hipError_t status = hipMalloc(pointerToAllocatedMemory, memorySizeInBytes);
	if (status != hipSuccess) {
		throw std::runtime_error(
				"allocateGpuMemory call failed with the CUDA error code " + CudaUtils::toErrorDescription(status)
		);
	}
}

[[maybe_unused]] __host__ void CudaUtils::freeGpuMemory(void **gpuMemoryPointer) {
	const hipError_t status = hipFree(gpuMemoryPointer);
	if (status != hipSuccess) {
		throw std::runtime_error(
				"freeGpuMemory call failed with the CUDA error code " + CudaUtils::toErrorDescription(status)
		);
	}

}

[[maybe_unused]] __host__ void CudaUtils::copyDataFromCpuMemoryToGpuMemory(
		const void *sourceCpuMemory,
		void *destinationGpuMemory,
		const size_t numBytesToCopy
) {
	copyDataBetweenCpuAndGpu(destinationGpuMemory, sourceCpuMemory, numBytesToCopy, hipMemcpyHostToDevice);
}

[[maybe_unused]] __host__ void CudaUtils::copyDataFromGpuMemoryToCpuMemory(
		const void *sourceGpuMemory,
		void *destinationCpuMemory,
		const size_t numBytesToCopy
) {
	copyDataBetweenCpuAndGpu(destinationCpuMemory, sourceGpuMemory, numBytesToCopy, hipMemcpyDeviceToHost);
}