#include "error.cuh"

std::string CudaUtils::toErrorDescription(const hipError_t cudaErrorCode) {
	switch (cudaErrorCode) {
		case hipSuccess:
			return "'hipSuccess'. The API call returned with no errors.";
		case hipErrorInvalidValue:
			return "'hipErrorInvalidValue'. "
				   "This indicates that one or more of the parameters passed to the API call is not within an acceptable range of values.";
		case hipErrorOutOfMemory:
			return "'hipErrorOutOfMemory'. "
				   "The API call failed because it was unable to allocate enough memory to perform the requested operation.";
		case hipErrorNotInitialized: // = 3
			return "'hipErrorNotInitialized'. "
				   "The API call failed because the CUDA driver and runtime could not be initialized.";
		case hipErrorInvalidMemcpyDirection: // = 21
			return "'hipErrorInvalidMemcpyDirection.' "
				   "This indicates that the direction of the memcpy passed to the API call is not one of the types specified by ::hipMemcpyKind.";
		case hipErrorInsufficientDriver: // = 35
			return "' hipErrorInsufficientDriver'. "
				   "This indicates that the installed NVIDIA CUDA driver is older than the CUDA runtime library. "
				   "Users should install an updated NVIDIA display driver to allow the application to run.";
		case hipErrorNoDevice: // = 100
			return "'hipErrorNoDevice'. "
				   "This indicates that no CUDA-capable devices were detected by the installed CUDA driver.";
		case cudaErrorNotPermitted: // = 800
			return "'cudaErrorNotPermitted. '"
				   "This error indicates the attempted operation is not permitted.";
		default:
			return "Unknown error code: " + std::to_string(cudaErrorCode);
	}
}