#include "gpu_facade.cuh"

using namespace CudaToolkit;

namespace {
	std::string getErrorDescription(const hipError_t errorCode) {
		switch (errorCode) {
			case hipErrorInvalidValue:
				return "'hipErrorInvalidValue': "
					   "This indicates that one or more of the parameters passed to the API call is not within an acceptable range of values.";
			case hipErrorOutOfMemory:
				return "'hipErrorOutOfMemory': "
					   "The API call failed because it was unable to allocate enough memory to perform the requested operation.";
			case hipErrorNotInitialized: // = 3
				return "'hipErrorNotInitialized'. "
					   "The API call failed because the CUDA driver and runtime could not be initialized.";
			case hipErrorInvalidMemcpyDirection: // = 21
				return "'hipErrorInvalidMemcpyDirection.' "
					   "This indicates that the direction of the memcpy passed to the API call is not one of the types specified by ::hipMemcpyKind.";
			case hipErrorInsufficientDriver: // = 35
				return "' hipErrorInsufficientDriver': "
					   "This indicates that the installed NVIDIA CUDA driver is older than the CUDA runtime library. "
					   "Users should install an updated NVIDIA display driver to allow the application to run.";
			case hipErrorNoDevice: // = 100
				return "'hipErrorNoDevice': "
					   "This indicates that no CUDA-capable devices were detected by the installed CUDA driver.";
			case cudaErrorNotPermitted: // = 800
				return "'cudaErrorNotPermitted': "
					   "This error indicates the attempted operation is not permitted.";
			default:
				return "'" + std::to_string(errorCode) + "': Unknown error.";
		}
	}
}


GpuFacade::GpuFacade() : gpuUsedByCurrentProcess(false) {

}

GpuFacade::~GpuFacade() {
	if (gpuUsedByCurrentProcess) {
		// Ensure the GPU is reset on program termination
		hipDeviceReset();
		gpuUsedByCurrentProcess = false;
	}
}

[[maybe_unused]] GpuFacade &GpuFacade::GpuFacade::getInstance() {
	static GpuFacade instance;
	return instance;
}

void GpuFacade::copyDataBetweenCpuAndGpu(
		void *destinationGpuMemory,
		const void *sourceCpuMemory,
		const size_t numBytesToCopy,
		const hipMemcpyKind copyDirection
) {
	const hipError_t status = hipMemcpy(
			destinationGpuMemory,
			sourceCpuMemory,
			numBytesToCopy,
			copyDirection
	);
	if (status) {
		throw std::runtime_error("hipMemcpy call failed with error code " + getErrorDescription(status));
	} else {
		gpuUsedByCurrentProcess = true;
	}
}
