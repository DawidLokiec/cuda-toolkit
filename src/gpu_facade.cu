#include "gpu_facade.cuh"

using namespace CudaToolkit;

GpuFacade::GpuFacade() : gpuUsedByCurrentProcess_(false) {

}

GpuFacade::~GpuFacade() {
	if (gpuUsedByCurrentProcess_) {
		// Ensure the GPU is reset on program termination
		hipDeviceReset();
		gpuUsedByCurrentProcess_ = false;
	}
}

[[maybe_unused]] GpuFacade &GpuFacade::GpuFacade::getInstance() {
	static GpuFacade instance;
	return instance;
}


std::string GpuFacade::getErrorDescription(const hipError_t errorCode) {
	switch (errorCode) {
		case hipErrorInvalidValue: // = 1
			return "'hipErrorInvalidValue': "
				   "This indicates that one or more of the parameters passed to the API call is not within an acceptable "
				   "range of values.";
		case hipErrorOutOfMemory: // = 2
			return "'hipErrorOutOfMemory': "
				   "The API call failed because it was unable to allocate enough memory to perform the requested "
				   "operation.";
		case hipErrorNotInitialized: // = 3
			return "'hipErrorNotInitialized': "
				   "The API call failed because the CUDA driver and runtime could not be initialized.";
		case hipErrorInvalidMemcpyDirection: // = 21
			return "'hipErrorInvalidMemcpyDirection': "
				   "This indicates that the direction of the memcpy passed to the API call is not one of the types "
				   "specified by hipMemcpyKind.";
		case hipErrorInsufficientDriver: // = 35
			return "' hipErrorInsufficientDriver': "
				   "This indicates that the installed NVIDIA CUDA driver is older than the CUDA runtime library. "
				   "Users should install an updated NVIDIA display driver to allow the application to run.";
		case hipErrorNoDevice: // = 100
			return "'hipErrorNoDevice': "
				   "This indicates that no CUDA-capable devices were detected by the installed CUDA driver.";
		case cudaErrorNotPermitted: // = 800
			return "'cudaErrorNotPermitted': "
				   "This error indicates the attempted operation is not permitted.";
		default:
			return "'" + std::to_string(errorCode) + "': Unknown error code.";
	}
}
